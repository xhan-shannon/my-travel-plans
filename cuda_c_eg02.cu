#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DATA_SIZE 1048576

int data[DATA_SIZE];

void generateNumbers(int *number, int size)
{
    for(int i = 0; i < size; i++){
        number[i] = rand() % 100;
    }
}

__global__ static void sumOfSquares(int *num, int *result)
{
    int sum = 0;
    int i;
    for(i = 0; i < DATA_SIZE; i++){
        sum += num[i] * num[i];
    }

    *result = sum;
}

int main(){
    generateNumbers(data, DATA_SIZE);
    
    int *gpudata;
    int *result;
    hipMalloc((void**)&gpudata, size(int) * DATA_SIZE);
    hipMalloc((void**)&result, size(int));

    hipMemcpy(gpudata, data, sizeof(int)*DATA_SIZE, hipMemcpyHostToDevice);
    sumOfSquare<<<1, 1, 0>>>(gpudata, result);

    int sum;
    hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(gpudata);
    hipFree(result);

    return 0;
}
